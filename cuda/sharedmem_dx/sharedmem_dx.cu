#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <conio.h>
#include "libcode.h"

__global__ void mykernel() 
{
	int idx = threadIdx.x;
	printf("gpu_sbin[%d] = %d\n", idx, gpu_sbin[idx]);
}

void myPrintGpuInfo()
{
	int count = 0;
	hipError_t cuerr = hipGetDeviceCount( &count );
	if(count==0)
	{
		printf("No NVIDIA GPU found. This program cannot continue.\n");
		exit(2);
	}

	int cudaver = 0;
	hipRuntimeGetVersion(&cudaver);
	hipDeviceProp_t  prop = {};
	hipGetDeviceProperties( &prop, 0 );
	printf("GPU: %s, CUDA ver: %d\n", prop.name, cudaver);
}


int main(int argc, char* argv[])
{
	int threads = BIN256;
	if(argc>1)
		threads = strtoul(argv[1], nullptr, 0);

	printf("Hello, sharedmem_dx!\n");

	myPrintGpuInfo();

	mykernel<<<1, threads>>>();

	hipError_t kerr = hipDeviceSynchronize();
	if(kerr)
		printf("CUDA kernel call error(%d): %s\n", kerr, hipGetErrorName(kerr));
	else
		printf("CUDA kernel call OK.\n");

	printf("Done. (press any key)\n");
	_getch();
}
